#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <hipblas.h>

#define DIM_BLOCK 128
#define SHARED_POINTS 100

__host__ __device__
inline float dist_square(int dimension, int num_points, float *points, int obj_idx, float *p2) {
	float ans = 0.0, tmp;
	float coord, coord_next;

	// prefetch
	coord_next = points[obj_idx];

	for (int i = 0; i < dimension; i++) {
		coord = coord_next;
		coord_next = points[(i + 1) * num_points + obj_idx];  // transposed
		tmp = coord - p2[i];
		ans += tmp * tmp;
	}
	return ans;
}

__global__
void nearest_cluster_new(float *point_norm, float *cluster_norm, float *pc_product,
int num_points, int num_coords, int num_clusters, float *new_clusters, 
int *membership, int *membership_changes, int *clusters_size, float *points) {

  unsigned int gdim = gridDim.x;
	unsigned int bid = blockIdx.x;
	unsigned int bdim = blockDim.x;
	unsigned int tid = threadIdx.x;
  unsigned int num_threads = gdim * bdim;
  unsigned int obj_idx;

	extern __shared__ float shared[];
	float *s_cluster_norm = shared;

	__syncthreads();

  int s_per_time = SHARED_POINTS;
  int times = (int) num_clusters / s_per_time;

  int new_cluster_idx = 0;
  float dist, min_dist = 3.40282e+38;

  for (obj_idx = bid * bdim + tid; obj_idx < num_points; obj_idx += num_threads) {

    // save centroid norms into shared memory by tiles, and calculate distances
    for (int t = 0; t < times; t ++) {

      for (int i = tid; i < s_per_time; i += bdim) {
        s_cluster_norm[i] = cluster_norm[t * s_per_time + i];
      }

	    __syncthreads();

#ifdef DEVICE_TIMING
clock_t start;
clock_t duration;
if (tid == 0) { start = clock(); }
#endif

      for (int i = 0; i < s_per_time; i++) {
      if ((dist = point_norm[obj_idx] + s_cluster_norm[i] - 
               pc_product[i * num_points + obj_idx]) < min_dist) {
				  min_dist = dist;
				  new_cluster_idx = i;
	  		}
		  }
    }

    // process the remaining clusters.
    // The final loop is unrolled to avoid an extra comparison in the previous loops
    for (int t = times * s_per_time; t < num_clusters; t ++) {

      for (int i = tid; i < num_clusters - times * s_per_time; i += bdim) {
        s_cluster_norm[i] = cluster_norm[times * s_per_time + i];
      }

      __syncthreads();

      for (int i = 0; i < s_per_time; i++) {
      if ((dist = point_norm[obj_idx] + s_cluster_norm[i] - 
               pc_product[i * num_points + obj_idx]) < min_dist) {
				  min_dist = dist;
				  new_cluster_idx = i;
	  		}
		  }
    }

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tdist time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

    int old_cluster_idx = membership[obj_idx];
	  #ifdef SYNCOUNT
  	membership_changes[bid] = __syncthreads_count(old_cluster_idx != new_cluster_idx);
  	#endif
	  if (old_cluster_idx != new_cluster_idx) {
		  #ifndef SYNCOUNT
    	atomicAdd(membership_changes, 1);
	  	#endif
		  membership[obj_idx] = new_cluster_idx;
    }

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tmemb cal time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

  	atomicAdd(&clusters_size[new_cluster_idx], 1);
	  for (int i = 0; i < num_coords; i++) {
		  atomicAdd(&new_clusters[new_cluster_idx * num_coords + i], 
            points[i * num_points + obj_idx]);
  	}

  	__syncthreads();

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tcent cal time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

  }
}

__global__
void nearest_cluster(float *points, float *clusters, int num_points, int num_coords, int num_clusters,
		float *new_clusters, int *membership, int *membership_changes, int *clusters_size) {

	unsigned int bid = blockIdx.x;
	unsigned int bdim = blockDim.x;
	unsigned int tid = threadIdx.x;
  unsigned int obj_idx = bid * bdim + tid;

	extern __shared__ float shared[];
	float *s_clusters = shared;

	__syncthreads();

  int s_per_time = (int) (SHARED_POINTS / (num_coords));
  int length_per_time = s_per_time * num_coords;
  int times = (int) num_clusters / s_per_time;

  int new_cluster_idx = 0;
  float dist, min_dist = 3.40282e+38;

  // save centroids into shared memory by tiles, and calculate distances
  for (int t = 0; t < times; t ++) {

    for (int i = tid; i < length_per_time; i += bdim) {
      s_clusters[i] = clusters[t * s_per_time + i];
    }

	  __syncthreads();

#ifdef DEVICE_TIMING
clock_t start;
clock_t duration;
if (tid == 0) { start = clock(); }
#endif

	  if (obj_idx < num_points) {

      for (int i = 0; i < s_per_time; i++) {
      if ((dist = dist_square(num_coords, num_points, points, obj_idx, 
            &s_clusters[i * num_coords]))
			  		< min_dist) {
				  min_dist = dist;
				  new_cluster_idx = i;
	  		}
		  }
    }
  }

  // process the remaining clusters.
  // The final loop is unrolled to avoid an extra comparison in the previous loops
  for (int t = times * s_per_time; t < num_clusters; t ++) {

    for (int i = tid; i < (num_clusters - times * s_per_time) * num_coords; 
             i += bdim) {
      s_clusters[i] = clusters[times * s_per_time + i];
    }

    __syncthreads();

    if (obj_idx < num_points) {

      for (int i = 0; i < num_clusters - times * s_per_time; i++) {
      if ((dist = dist_square(num_coords, num_points, points, obj_idx, 
            &s_clusters[i * num_coords]))
            < min_dist) {
          min_dist = dist;
          new_cluster_idx = i;
        }
      }
    }

  }

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tdist time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

  if (obj_idx < num_points) {
    int old_cluster_idx = membership[obj_idx];
	  #ifdef SYNCOUNT
  	membership_changes[bid] = __syncthreads_count(old_cluster_idx != new_cluster_idx);
  	#endif
	  if (old_cluster_idx != new_cluster_idx) {
		  #ifndef SYNCOUNT
    	atomicAdd(membership_changes, 1);
	  	#endif
		  membership[obj_idx] = new_cluster_idx;
    }

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tmemb cal time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

  	atomicAdd(&clusters_size[new_cluster_idx], 1);
	  for (int i = 0; i < num_coords; i++) {
		  atomicAdd(&new_clusters[new_cluster_idx * num_coords + i], 
            points[i * num_points + obj_idx]);
  	}
  }

	__syncthreads();

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tcent cal time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

}

inline void checkCudaError(hipError_t error) {
	if (error != hipSuccess)
	{
		printf("cuda error code %d: %s\n", error, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

inline void checkCudaError( int line, hipError_t error) {
  if (error != hipSuccess)
  {
    printf("cuda error code %d, line(%d): %s\n", error, line, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

float **kmeans(float **points, int num_points, int num_coords, int num_clusters,
			float threshold, int iterations, int *membership) {

	// initialization
	int i, j;
	int *clusters_size;
	size_t points_length = num_points * num_coords * sizeof(float);
	size_t clusters_length = num_clusters * num_coords * sizeof(float);
  size_t pc_product_length = num_points * num_clusters * sizeof(float);
	float **retval, **clusters;
  float **trans_points;
  float **trans_clusters;
  float *pc_product;
  float *point_norm;
  float *cluster_norm;

	// transpose the points to coalesce
  trans_points = (float**) malloc(num_coords * sizeof(float*));
  assert(trans_points);
  trans_points[0] = (float*) malloc(points_length);
  assert(trans_points[0]);
  for (i = 0; i < num_coords; i++) {
    if ( i > 0) trans_points[i] = trans_points[i - 1] + num_points;
		for (j = 0; j < num_points; j++)
			trans_points[i][j] = points[j][i];
	}

	retval = (float**) malloc(num_clusters * sizeof(float*));
	assert(retval);
	retval[0] = (float*) malloc(clusters_length);
	assert(retval[0]);
	for (i = 1; i < num_clusters; i++) {
		retval[i] = retval[i - 1] + num_coords;
	}

  trans_clusters = (float**) malloc(num_coords * sizeof(float*));
  assert(trans_clusters);
  trans_clusters[0] = (float*) malloc(clusters_length);
  assert(trans_clusters[0]);

  pc_product = (float*) calloc(num_points * num_clusters, sizeof(float));

  point_norm = (float*) malloc(num_points * sizeof(float));
  cluster_norm = (float*) malloc(num_clusters * sizeof(float));

	// randomly choose initial clusters
	for (i=0; i < num_clusters; i++) {
    		memcpy(retval[i], points[i], num_coords * sizeof(float));
	}

	memset (membership, -1, sizeof(membership));

	// allocate space for temp clusters
	clusters_size = (int*) calloc(num_clusters, sizeof(int));
	assert(clusters_size);

	clusters = (float**) malloc(num_clusters * sizeof(float*));
	assert(clusters);
	clusters[0] = (float*) calloc(num_clusters * num_coords, sizeof(float));
	assert(clusters[0]);
	for (i = 1; i < num_clusters; i++) {
		clusters[i] = clusters[i - 1] + num_coords;
	}

  // Prepare computation kernel

	const unsigned int dimBlock = DIM_BLOCK;
	const unsigned int dimGrid = (num_points - 1) / dimBlock + 1;

	#ifdef SYNCOUNT
	int *tmp_membership_changes = (int*) calloc(dimGrid, sizeof(int));
	assert(tmp_membership_changes);
	#endif

	// Cuda device memory allocation
	float *device_points, *device_trans_points;
	float *device_clusters, *device_trans_clusters;
	float *device_new_clusters;
  float *device_pc_product;
  float *device_point_norm, *device_cluster_norm;
  float *d_vector;
	int *device_membership;
	int *device_membership_changes;
	int *device_clusters_size;
  float alpha = 2.0f;
  float beta = 0.0f;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

	checkCudaError(__LINE__, hipMalloc(&device_points, points_length));
	checkCudaError(__LINE__, hipMalloc(&device_trans_points, points_length));
	checkCudaError(__LINE__, hipMalloc(&device_clusters, clusters_length));
	checkCudaError(__LINE__, hipMalloc(&device_trans_clusters, clusters_length));
	checkCudaError(__LINE__, hipMalloc(&device_new_clusters, clusters_length));
	checkCudaError(__LINE__, hipMalloc(&device_pc_product, pc_product_length));
  checkCudaError(__LINE__, hipMalloc(&device_point_norm, num_points * sizeof(float)));
  checkCudaError(__LINE__, hipMalloc(&device_cluster_norm, num_clusters * sizeof(float)));
  checkCudaError(__LINE__, hipMalloc(&d_vector, num_coords * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_clusters_size, num_clusters * sizeof(int)));
	checkCudaError(__LINE__, hipMalloc(&device_membership, num_points * sizeof(int)));
	checkCudaError(__LINE__, hipMalloc(&device_membership_changes, dimGrid * sizeof(int)));

//	checkCudaError(__LINE__, hipMemcpy(device_points, points[0],
//      points_length, hipMemcpyHostToDevice));
	checkCudaError(__LINE__, hipMemcpy(device_trans_points, trans_points[0],
      points_length, hipMemcpyHostToDevice));
	checkCudaError(__LINE__, hipMemcpy(device_membership,
			membership, num_points * sizeof(int), hipMemcpyHostToDevice));

  stat = hipblasCreate(&handle);

  for (i = 0; i < num_points; i ++) {
    stat = hipblasSetVector(num_coords, sizeof(float), points[i], 1, d_vector, 1);
    stat = hipblasSnrm2(handle, num_coords, d_vector, 1, &point_norm[i]);
    point_norm[i] = point_norm[i] * point_norm[i];
  }

	// K-mean calculation
	int iter = 0;
	int membership_changes = num_points;
	
	while (((float) membership_changes / (float) num_points > threshold) && (iter++ < iterations)) {

		DEBUG_LOG("iteration=%d, threshold=%5.5f\n", iter, (float) membership_changes / (float) num_points);
		membership_changes = 0;

#ifdef KERNAL_TIMING
int64_t start = GetTimeMius64();
#endif


		// initialize
		#ifndef SYNCOUNT
    checkCudaError(__LINE__, hipMemcpy(device_membership_changes, &membership_changes,
        sizeof(int), hipMemcpyHostToDevice));
		#endif

  	// transpose the clusters to coalesce
    for (i = 0; i < num_coords; i++) {
      if ( i > 0) trans_clusters[i] = trans_clusters[i - 1] + num_clusters;
		  for (j = 0; j < num_clusters; j++)
			  trans_clusters[i][j] = retval[j][i];
	  }

//    checkCudaError(__LINE__, hipMemset(device_pc_product, 0, 
//        pc_product_length));
		checkCudaError(__LINE__, hipMemcpy(device_clusters, retval[0],
				clusters_length, hipMemcpyHostToDevice));
//  	checkCudaError(__LINE__, hipMemcpy(device_trans_clusters, trans_clusters[0], 
//        clusters_length, hipMemcpyHostToDevice));
		checkCudaError(__LINE__, hipMemcpy(device_new_clusters, clusters[0],
				clusters_length, hipMemcpyHostToDevice));
		checkCudaError(__LINE__, hipMemcpy(device_clusters_size, clusters_size,
				num_clusters * sizeof(int), hipMemcpyHostToDevice));

#ifdef KERNAL_TIMING
int64_t duration = GetTimeMiusFrom(start);
printf("prep time = %lld microseconds\n", (long long) duration);
start = GetTimeMius64();
#endif

    memset (pc_product, 0, pc_product_length);

    // (x_i - c_j)^2 = (x_i)^2 + (c_j)^2 - 2*x_i*c_j
    // 1. Use cuBLAS to compute x_i*c_j
    stat = hipblasSetMatrix(num_clusters, num_coords, sizeof(float), trans_clusters, num_clusters, device_trans_clusters, num_clusters);
    stat = hipblasSetMatrix(num_coords, num_points, sizeof(float), points[0], num_coords, device_points, num_coords);
    stat = hipblasSetMatrix(num_clusters, num_points, sizeof(float), pc_product, num_clusters, device_pc_product, num_clusters);

		hipDeviceSynchronize();

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_clusters, num_points, 
              num_coords, &alpha, device_trans_clusters, num_clusters, 
              device_points, num_coords, &beta, device_pc_product, num_clusters);

		hipDeviceSynchronize();

    // 2. Compute (x_i)^2 and (c_j)^2
    for (i = 0; i < num_clusters; i ++) {
      stat = hipblasSetVector(num_coords, sizeof(float), retval[i], 1, d_vector, 1);
      stat = hipblasSnrm2(handle, num_coords, d_vector, 1, &cluster_norm[i]);
      cluster_norm[i] = cluster_norm[i] * cluster_norm[i];
    }

		hipDeviceSynchronize();

  	checkCudaError(__LINE__, hipMemcpy(device_point_norm, point_norm,
				num_points * sizeof(float), hipMemcpyHostToDevice));
  	checkCudaError(__LINE__, hipMemcpy(device_cluster_norm, cluster_norm,
				num_clusters * sizeof(float), hipMemcpyHostToDevice));

    // 3. Compute nearest cluster

    nearest_cluster_new
				<<<dimGrid, dimBlock, SHARED_POINTS * sizeof(float)>>>
        (device_point_norm, device_cluster_norm, device_pc_product, 
         num_points, num_coords, num_clusters, device_new_clusters, 
         device_membership, device_membership_changes, device_clusters_size, 
         device_trans_points);

#if 0
		nearest_cluster
				<<<dimGrid, dimBlock, clusters_length>>>
        (device_trans_points, device_clusters, num_points, num_coords, num_clusters,
         device_new_clusters, device_membership, device_membership_changes, 
         device_clusters_size);
#endif

		hipDeviceSynchronize();
		checkCudaError(__LINE__, hipGetLastError());

#ifdef KERNAL_TIMING
duration = GetTimeMiusFrom(start);
printf("kernal time = %lld microseconds\n", (long long) duration);
start = GetTimeMius64();
#endif

		checkCudaError(__LINE__, hipMemcpy(clusters_size, device_clusters_size,
        num_clusters * sizeof(int), hipMemcpyDeviceToHost));
		checkCudaError(__LINE__, hipMemcpy(clusters[0], device_new_clusters,
				clusters_length, hipMemcpyDeviceToHost));
		checkCudaError(__LINE__, hipMemcpy(membership, device_membership,
				num_points * sizeof(int), hipMemcpyDeviceToHost));
		#ifdef SYNCOUNT
		checkCudaError(__LINE__, hipMemcpy(tmp_membership_changes, device_membership_changes,
        dimGrid * sizeof(int), hipMemcpyDeviceToHost));

		for (i = 0; i < dimGrid; i ++) {
			membership_changes += tmp_membership_changes[i];
		}
		#else
    checkCudaError(__LINE__, hipMemcpy(&membership_changes, device_membership_changes,
        sizeof(int), hipMemcpyDeviceToHost));
		#endif

#ifdef KERNAL_TIMING
duration = GetTimeMiusFrom(start);
printf("end cpy time = %lld microseconds\n", (long long) duration);
start = GetTimeMius64();
#endif

		// calculate new cluster centers
		for (i = 0; i < num_clusters; i++) {
			for (j = 0; j < num_coords; j++) {
				if (clusters_size[i] > 0) {
					retval[i][j] = clusters[i][j] / clusters_size[i];
				}
				clusters[i][j] = 0.0;
			}
			clusters_size[i] = 0;
		}

#ifdef KERNAL_TIMING
duration = GetTimeMiusFrom(start);
printf("centroid cal time = %lld microseconds\n", (long long) duration);
#endif

	}

	free(trans_points[0]);
  free(trans_points);
	free(clusters[0]);
	free(clusters);
  free(trans_clusters[0]);
  free(trans_clusters);
  free(pc_product);
  free(point_norm);
  free(cluster_norm);
	free(clusters_size);

  checkCudaError(__LINE__, hipFree(device_points));
	checkCudaError(__LINE__, hipFree(device_trans_points));
	checkCudaError(__LINE__, hipFree(device_clusters));
	checkCudaError(__LINE__, hipFree(device_trans_clusters));
	checkCudaError(__LINE__, hipFree(device_new_clusters));
  checkCudaError(__LINE__, hipFree(device_pc_product));
  checkCudaError(__LINE__, hipFree(d_vector));
  checkCudaError(__LINE__, hipFree(device_point_norm));
  checkCudaError(__LINE__, hipFree(device_cluster_norm));
	checkCudaError(__LINE__, hipFree(device_membership));
	checkCudaError(__LINE__, hipFree(device_membership_changes));
	checkCudaError(__LINE__, hipFree(device_clusters_size));

	#ifdef SYNCOUNT
	free(tmp_membership_changes);
	#endif

  hipblasDestroy(handle);

	return retval;
}
