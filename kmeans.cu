#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <hipblas.h>

#define DIM_BLOCK 128

__host__ __device__
inline float dist_square(int dimension, int num_points, float *points, int obj_idx, float *p2) {
	float ans = 0.0, tmp;
	float coord, coord_next;

	// prefetch
	coord_next = points[obj_idx];

	for (int i = 0; i < dimension; i++) {
		coord = coord_next;
		coord_next = points[(i + 1) * num_points + obj_idx];  // transposed
		tmp = coord - p2[i];
		ans += tmp * tmp;
	}
	return ans;
}

__global__
void nearest_cluster(float *points, float *clusters, int num_points, int num_coords, int num_clusters,
		float *new_clusters, int *membership, int *membership_changes, int *clusters_size) {

	unsigned int bid = blockIdx.x;
	unsigned int bdim = blockDim.x;
	unsigned int tid = threadIdx.x;
	//unsigned int clusters_length = num_clusters * num_coords;
  unsigned int obj_idx = bid * bdim + tid;

	extern __shared__ float shared[];
	float *s_clusters = shared;

	__syncthreads();

  int s_per_time = (int) (100 / (num_coords));
  int length_per_time = s_per_time * num_coords;
  int times = (int) num_clusters / s_per_time;

  int new_cluster_idx = 0;
  float dist, min_dist = 3.40282e+38;

  // save centroids into shared memory by tiles, and calculate distances
  for (int t = 0; t < times; t ++) {

    for (int i = tid; i < length_per_time; i ++) {
      s_clusters[i] = clusters[t * s_per_time + i];
    }

	  __syncthreads();

#ifdef DEVICE_TIMING
clock_t start;
clock_t duration;
if (tid == 0) { start = clock(); }
#endif

	  if (obj_idx < num_points) {

      for (int i = 0; i < s_per_time; i++) {
      if ((dist = dist_square(num_coords, num_points, points, obj_idx, 
            &s_clusters[i * num_coords]))
			  		< min_dist) {
				  min_dist = dist;
				  new_cluster_idx = i;
	  		}
		  }
    }
  }

  // process the remaining clusters.
  // The final loop is unrolled to avoid an extra comparison in the previous loops
  for (int t = times * s_per_time; t < num_clusters; t ++) {

    for (int i = tid; i < (num_clusters - times * s_per_time) * num_coords; i ++) {
      s_clusters[i] = clusters[times * s_per_time + i];
    }

    __syncthreads();

    if (obj_idx < num_points) {

      for (int i = 0; i < num_clusters - times * s_per_time; i++) {
      if ((dist = dist_square(num_coords, num_points, points, obj_idx, 
            &s_clusters[i * num_coords]))
            < min_dist) {
          min_dist = dist;
          new_cluster_idx = i;
        }
      }
    }

  }

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tdist time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

  if (obj_idx < num_points) {
    int old_cluster_idx = membership[obj_idx];
	  #ifdef SYNCOUNT
  	membership_changes[bid] = __syncthreads_count(old_cluster_idx != new_cluster_idx);
  	#endif
	  if (old_cluster_idx != new_cluster_idx) {
		  #ifndef SYNCOUNT
    	atomicAdd(membership_changes, 1);
	  	#endif
		  membership[obj_idx] = new_cluster_idx;
    }

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tmemb cal time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

  	atomicAdd(&clusters_size[new_cluster_idx], 1);
	  for (int i = 0; i < num_coords; i++) {
		  atomicAdd(&new_clusters[new_cluster_idx * num_coords + i], 
            points[i * num_points + obj_idx]);
  	}
  }

	__syncthreads();

#ifdef DEVICE_TIMING
if (tid == 0) {
duration = clock() - start;
printf("\tcent cal time = %lld microseconds\n", (long long) duration);
start = clock();
}
#endif

}

inline void checkCudaError(hipError_t error) {
	if (error != hipSuccess)
	{
		printf("cuda error code %d: %s\n", error, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

inline void checkCudaError( int line, hipError_t error) {
  if (error != hipSuccess)
  {
    printf("cuda error code %d, line(%d): %s\n", error, line, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

float **kmeans(float **points, int num_points, int num_coords, int num_clusters,
			float threshold, int iterations, int *membership) {

	// initialization
	int i, j;
	float **retval, **clusters;
	int *clusters_size;
	unsigned int points_length = num_points * num_coords;
	unsigned int clusters_length = num_clusters * num_coords;

	// transpose the points to coalesce
  float **trans_points = (float**) malloc(num_coords * sizeof(float*));
  assert(trans_points);
  trans_points[0] = (float*) malloc(points_length * sizeof(float));
  assert(trans_points[0]);
  for (i = 0; i < num_coords; i++) {
    if ( i > 0) trans_points[i] = trans_points[i - 1] + num_points;
		for (j = 0; j < num_points; j++)
			trans_points[i][j] = points[j][i];
	}

	retval = (float**) malloc(num_clusters * sizeof(float*));
	assert(retval);
	retval[0] = (float*) malloc(clusters_length * sizeof(float));
	assert(retval[0]);
	for (i = 1; i < num_clusters; i++) {
		retval[i] = retval[i - 1] + num_coords;
	}

  float **trans_clusters = (float**) malloc(num_coords * sizeof(float*));
  assert(trans_clusters);
  trans_clusters[0] = (float*) malloc(clusters_length * sizeof(float));
  assert(trans_clusters[0]);

	// randomly choose initial clusters
	for (i=0; i < num_clusters; i++) {
    		memcpy(retval[i], points[i], num_coords * sizeof(float));
	}

	memset (membership, -1, sizeof(membership));

	// allocate space for temp clusters
	clusters_size = (int*) calloc(num_clusters, sizeof(int));
	assert(clusters_size);

	clusters = (float**) malloc(num_clusters * sizeof(float*));
	assert(clusters);
	clusters[0] = (float*) calloc(clusters_length, sizeof(float));
	assert(clusters[0]);
	for (i = 1; i < num_clusters; i++) {
		clusters[i] = clusters[i - 1] + num_coords;
	}

  // Prepare computation kernel

	const unsigned int dimBlock = DIM_BLOCK;
	const unsigned int dimGrid = (num_points - 1) / dimBlock + 1;

	#ifdef SYNCOUNT
	int *tmp_membership_changes = (int*) calloc(dimGrid, sizeof(int));
	assert(tmp_membership_changes);
	#endif

	// Cuda device memory allocation
	float *device_points, *device_trans_points;
	float *device_clusters, *device_trans_clusters;
	float *device_new_clusters;
  float *device_pc_product;
	int *device_membership;
	int *device_membership_changes;
	int *device_clusters_size;

  int pc_product_size = num_points * num_clusters;
  float alpha = 2.0f;
  float beta = 1.0f;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

	checkCudaError(__LINE__, hipMalloc(&device_points, points_length * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_trans_points, points_length * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_clusters, clusters_length * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_trans_clusters, clusters_length * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_new_clusters, clusters_length * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_pc_product, pc_product_size * sizeof(float)));
	checkCudaError(__LINE__, hipMalloc(&device_clusters_size, num_clusters * sizeof(int)));
	checkCudaError(__LINE__, hipMalloc(&device_membership, num_points * sizeof(int)));
	checkCudaError(__LINE__, hipMalloc(&device_membership_changes, dimGrid * sizeof(int)));

	checkCudaError(__LINE__, hipMemcpy(device_points, points[0],
      points_length * sizeof(float), hipMemcpyHostToDevice));
	checkCudaError(__LINE__, hipMemcpy(device_trans_points, trans_points[0],
      points_length * sizeof(float), hipMemcpyHostToDevice));
	checkCudaError(__LINE__, hipMemcpy(device_membership,
			membership, num_points * sizeof(int), hipMemcpyHostToDevice));

	// K-mean calculation
	int iter = 0;
	int membership_changes = num_points;
	
	while (((float) membership_changes / (float) num_points > threshold) && (iter++ < iterations)) {

		DEBUG_LOG("iteration=%d, threshold=%5.5f\n", iter, (float) membership_changes / (float) num_points);
		membership_changes = 0;

#ifdef KERNAL_TIMING
int64_t start = GetTimeMius64();
#endif


		// initialize
		#ifndef SYNCOUNT
    checkCudaError(__LINE__, hipMemcpy(device_membership_changes, &membership_changes,
        sizeof(int), hipMemcpyHostToDevice));
		#endif

  	// transpose the clusters to coalesce
    for (i = 0; i < num_coords; i++) {
      if ( i > 0) trans_clusters[i] = trans_clusters[i - 1] + num_clusters;
		  for (j = 0; j < num_clusters; j++)
			  trans_clusters[i][j] = retval[j][i];
	  }

    checkCudaError(__LINE__, hipMemset(device_pc_product, 0, 
        pc_product_size * sizeof(float)));
		checkCudaError(__LINE__, hipMemcpy(device_clusters, retval[0],
				clusters_length * sizeof(float), hipMemcpyHostToDevice));
  	checkCudaError(__LINE__, hipMemcpy(device_trans_clusters, trans_clusters[0], 
        clusters_length * sizeof(float), hipMemcpyHostToDevice));
		checkCudaError(__LINE__, hipMemcpy(device_new_clusters, clusters[0],
				clusters_length * sizeof(float), hipMemcpyHostToDevice));
		checkCudaError(__LINE__, hipMemcpy(device_clusters_size, clusters_size,
				num_clusters * sizeof(int), hipMemcpyHostToDevice));

#ifdef KERNAL_TIMING
int64_t duration = GetTimeMiusFrom(start);
printf("prep time = %lld microseconds\n", (long long) duration);
start = GetTimeMius64();
#endif


    // (x_i - c_j)^2 = (x_i)^2 + (c_j)^2 - 2*x_i*c_j
    // First use cuBLAS to compute x_i*c_j

#if 0
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_clusters, num_points, 
              num_coords, &alpha, device_trans_clusters, num_clusters, 
              device_points, num_coords, &beta, device_pc_product, num_clusters);
#endif

//   stat = hipblasGetMatrix (m,n, sizeof (*c) ,d_c ,m,c,m); // cp d_c - >c



		// call kernel function
		nearest_cluster
				<<<dimGrid, dimBlock, clusters_length * sizeof(float)>>>
        (device_trans_points, device_clusters, num_points, num_coords, num_clusters, device_new_clusters,
        device_membership, device_membership_changes, device_clusters_size);

		hipDeviceSynchronize();
		checkCudaError(__LINE__, hipGetLastError());

#ifdef KERNAL_TIMING
duration = GetTimeMiusFrom(start);
printf("kernal time = %lld microseconds\n", (long long) duration);
start = GetTimeMius64();
#endif

		checkCudaError(__LINE__, hipMemcpy(clusters_size, device_clusters_size,
        num_clusters * sizeof(int), hipMemcpyDeviceToHost));
		checkCudaError(__LINE__, hipMemcpy(clusters[0], device_new_clusters,
				clusters_length * sizeof(float), hipMemcpyDeviceToHost));
		checkCudaError(__LINE__, hipMemcpy(membership, device_membership,
				num_points * sizeof(int), hipMemcpyDeviceToHost));
		#ifdef SYNCOUNT
		checkCudaError(__LINE__, hipMemcpy(tmp_membership_changes, device_membership_changes,
        dimGrid * sizeof(int), hipMemcpyDeviceToHost));

		for (i = 0; i < dimGrid; i ++) {
			membership_changes += tmp_membership_changes[i];
		}
		#else
    checkCudaError(__LINE__, hipMemcpy(&membership_changes, device_membership_changes,
        sizeof(int), hipMemcpyDeviceToHost));
		#endif

#ifdef KERNAL_TIMING
duration = GetTimeMiusFrom(start);
printf("end cpy time = %lld microseconds\n", (long long) duration);
start = GetTimeMius64();
#endif

		// calculate new cluster centers
		for (i = 0; i < num_clusters; i++) {
			for (j = 0; j < num_coords; j++) {
				if (clusters_size[i] > 0) {
					retval[i][j] = clusters[i][j] / clusters_size[i];
				}
				clusters[i][j] = 0.0;
			}
			clusters_size[i] = 0;
		}

#ifdef KERNAL_TIMING
duration = GetTimeMiusFrom(start);
printf("centroid cal time = %lld microseconds\n", (long long) duration);
#endif

	}

	free(trans_points[0]);
  free(trans_points);
  free(trans_clusters[0]);
  free(trans_clusters);

  checkCudaError(__LINE__, hipFree(device_points));
	checkCudaError(__LINE__, hipFree(device_trans_points));
	checkCudaError(__LINE__, hipFree(device_clusters));
	checkCudaError(__LINE__, hipFree(device_trans_clusters));
	checkCudaError(__LINE__, hipFree(device_new_clusters));
  checkCudaError(__LINE__, hipFree(device_pc_product));
	checkCudaError(__LINE__, hipFree(device_membership));
	checkCudaError(__LINE__, hipFree(device_membership_changes));
	checkCudaError(__LINE__, hipFree(device_clusters_size));

	free(clusters[0]);
	free(clusters);
	free(clusters_size);

	#ifdef SYNCOUNT
	free(tmp_membership_changes);
	#endif

	return retval;
}
